#include "layer.h"

Layer* GetLayer(NNParameter* nn_para, LayerParameter* layer_para)  {
  if (layer_para->type_ == "conv") {   
    return new ConvLayer(nn_para, layer_para);
  } else if (layer_para->type_ == "full_rect") {   
    return new FullRectLayer(nn_para, layer_para);  
  } else if (layer_para->type_ == "full_linear") {   
    return new FullLinearLayer(nn_para, layer_para);  
  } else {
    cerr << "no each layer" << endl;
    return NULL;
  }
}      
        
void ConvLayer::Init(Layer* bottom, Layer* top, hiprandGenerator_t gen, hipblasHandle_t handle) {    
  if (bottom == NULL) {
    input_ = NULL;      
    input_im_size_ = nn_para_->im_size_;
    input_size_ = input_im_size_*input_im_size_;
    input_num_ = 4;
  } else if (bottom != NULL) {
    input_ = bottom->output_;
    if (input_ != bottom->output_) {
	    cerr << "Init input == bottom output";
      exit(0);
    }

    input_im_size_ = bottom->output_im_size_;    
    input_size_ = bottom->output_size_;  
    input_num_ = bottom->output_num_;    
  }

  bottom_ = bottom;
  top_ = top;

  gen_ = gen;
  handle_ = handle;

  output_im_size_ = (input_im_size_ - filter_size_) / stride_ + 1;
  output_size_ = output_im_size_*output_im_size_;  
  hipMalloc(&input_, filter_num_*input_size_*sizeof(float));
  hipMalloc(&output_, filter_num_*output_size_*sizeof(float));
  hipMalloc(&delta_, filter_num_*output_size_*sizeof(float));
  hipMalloc(&weight_, input_num_*filter_num_*filter_size_*filter_size_*sizeof(float));
  hipMalloc(&bias_, filter_num_*sizeof(float));
  hipMalloc(&grad_weight_, input_num_*filter_num_*filter_size_*filter_size_*sizeof(float));
  hipMalloc(&grad_bias_, filter_num_*sizeof(float));
  hipMalloc(&im_col_, output_im_size_*output_im_size_*(filter_size_+1)*(filter_size_+1)*stride_*sizeof(float));
  hipMalloc(&ones_, output_im_size_*output_im_size_*sizeof(float));

  weight_cpu_ = (float*) malloc(input_num_*filter_num_*filter_size_*filter_size_*sizeof(float));
  bias_cpu_ = (float*) malloc(filter_num_*sizeof(float));

  RandGauss(weight_, input_num_*filter_num_*filter_size_*filter_size_, 0.001, gen_, handle_);
  Constants(bias_, filter_num_, 0.0);        
  Zeros(delta_, filter_num_*output_size_);
  Constants(ones_, output_im_size_*output_im_size_, 1.0);

}

void ConvLayer::Forward() {    
  Zeros(output_, filter_num_*output_size_);

  for (int i = 0; i < input_num_; ++i) {
    im2col_gpu(input_ + i*input_im_size_*input_im_size_, 1, input_im_size_, input_im_size_, filter_size_, 0, stride_, im_col_);

    float alpha = 1.0;
    float beta = 1.0;
    hipblasSgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N,
      output_im_size_*output_im_size_, filter_num_, filter_size_*filter_size_, 
      &alpha, im_col_, output_im_size_*output_im_size_, weight_ + i*filter_num_*filter_size_*filter_size_, filter_size_*filter_size_, 
      &beta, output_, output_im_size_*output_im_size_);
  }
 
  for (int i = 0; i < filter_num_; ++i) {
    float alpha = 1.0;
    float beta = 1.0;
    hipblasSgemv(handle_, HIPBLAS_OP_N, 1, output_im_size_*output_im_size_, 
      &alpha, ones_, 1, &bias_[i], 1, &beta, output_ + i*output_im_size_*output_im_size_, 1);
  }

  Rect(output_, filter_num_*output_size_);
}

void ConvLayer::Backward() {  
  if (bottom_ == NULL) {
    return;
  }

  for (int i = 0; i < input_num_; ++i) {
    Zeros(im_col_, output_im_size_*output_im_size_*filter_size_*filter_size_);

    float alpha = 1.0;
    float beta = 1.0; 
    hipblasSgemm(handle_,HIPBLAS_OP_N, HIPBLAS_OP_T, 
                output_im_size_*output_im_size_, filter_size_*filter_size_, filter_num_,
                &alpha, delta_, output_im_size_*output_im_size_, weight_ + i*filter_num_*filter_size_*filter_size_, filter_size_*filter_size_, 
		            &beta, im_col_, output_im_size_*output_im_size_);  

    col2im_gpu(im_col_, 1, input_im_size_, input_im_size_, filter_size_, 0,  stride_, bottom_->delta_ + i*input_im_size_*input_im_size_);        
  }

  // applying the deriative of rectifier function
  RectDiff(bottom_->output_, bottom_->delta_, input_num_*input_im_size_*input_im_size_);
}  

void ConvLayer::GetGradient() {
  Zeros(grad_weight_, input_num_*filter_num_*filter_size_*filter_size_);        
  Zeros(grad_bias_, filter_num_);        

  // Convolute input_ with delta_  
  int CONV_DIM = filter_size_ + 1;
  int FILTER_SIZE = output_im_size_;
  for (int i = 0; i < input_num_; ++i) {

    im2col_gpu(input_ + i*input_im_size_*input_im_size_, 1, input_im_size_, input_im_size_, FILTER_SIZE, 0, stride_, im_col_);  

    float alpha = 1.0;
    float beta = 1.0;
    hipblasSgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, 
              CONV_DIM*CONV_DIM, filter_num_, FILTER_SIZE*FILTER_SIZE,
              &alpha, im_col_, CONV_DIM*CONV_DIM, delta_, 
              FILTER_SIZE*FILTER_SIZE, &beta, grad_weight_ + i*filter_num_*filter_size_*filter_size_, CONV_DIM*CONV_DIM);
  }

  // grad_bc2
  for (int i = 0; i < filter_num_; ++i) {
    float alpha = 1.0;
    float beta = 1.0;
    hipblasSgemv(handle_, HIPBLAS_OP_T, output_im_size_*output_im_size_, 1, &alpha, 
                delta_ + i*output_im_size_*output_im_size_, output_im_size_*output_im_size_, ones_, 1, &beta, &grad_bias_[i], 1);          
  }

}

void ConvLayer::Update() {
  float alpha = - nn_para_->learn_rate_;
  hipblasSaxpy(handle_, input_num_*filter_num_*filter_size_*filter_size_, &alpha, grad_weight_, 1, weight_, 1);    
  hipblasSaxpy(handle_, filter_num_, &alpha, grad_bias_, 1, bias_, 1);    
}

void ConvLayer::WriteWeights(ofstream &file_out) {
  hipMemcpy(weight_cpu_, weight_, input_num_*filter_num_*filter_size_*filter_size_*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(bias_cpu_, bias_, filter_num_*sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < input_num_*filter_num_*filter_size_*filter_size_; ++i) {
    file_out << weight_cpu_[i] << " ";
  }

  for (int i = 0; i < filter_num_; ++i) {
    file_out << bias_cpu_[i] << " ";
  }
}

void ConvLayer::ReadWeights(ifstream &file_in) {
  cout << "ConvLayer read weights" << endl;
  for (int i = 0; i < input_num_*filter_num_*filter_size_*filter_size_; ++i) {
    file_in >> weight_cpu_[i];
  }

  for (int i = 0; i < filter_num_; ++i) {
    file_in >> bias_cpu_[i];
  }
  
  hipMemcpy(weight_ , weight_cpu_, input_num_*filter_num_*filter_size_*filter_size_*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(bias_, bias_cpu_, filter_num_*sizeof(float), hipMemcpyHostToDevice);
}

void FullRectLayer::Init(Layer* bottom, Layer* top, hiprandGenerator_t gen, hipblasHandle_t handle) {
  if (bottom == NULL) {
    input_ = NULL;      
    input_im_size_ = nn_para_->im_size_;
    input_size_ = input_im_size_*input_im_size_;
    input_num_ = 1;
  } else {
    input_ = bottom->output_;
    input_im_size_ = bottom->output_im_size_;      
    input_size_ = bottom->output_size_;  
    input_num_ = bottom->output_num_;    
  }

  bottom_ = bottom;
  top_ = top;

  gen_ = gen;
  handle_ = handle;

  output_im_size_ = 0;
  output_num_ = 1;    

  hipMalloc(&output_, output_size_*sizeof(float));
  hipMalloc(&delta_, output_size_*sizeof(float));
  hipMalloc(&weight_, input_num_*input_num_*input_size_*output_size_*sizeof(float));
  hipMalloc(&bias_, output_size_*sizeof(float));
  hipMalloc(&grad_weight_, input_num_*input_size_*output_size_*sizeof(float));
  hipMalloc(&grad_bias_, output_size_*sizeof(float));

  weight_cpu_ = (float*)malloc(input_num_*input_num_*input_size_*output_size_*sizeof(float));
  bias_cpu_ = (float*)malloc(output_size_*sizeof(float));

  RandGauss(weight_, input_num_*input_num_*input_size_*output_size_, 0.001, gen_, handle_);
  Constants(bias_, output_size_, 0.0);
  Zeros(delta_, output_size_);

  
}

void FullRectLayer::Forward() {
  Zeros(output_, output_size_);        
  float alpha = 1.0;
  float beta = 0.0;
  hipblasSgemv(handle_, HIPBLAS_OP_T, input_num_*input_size_, output_size_, &alpha, 
              weight_, input_num_*input_size_, input_, 1, &beta, output_, 1);   

  hipblasSaxpy(handle_, output_size_, &alpha, bias_, 1, output_, 1);    
  Rect(output_, output_size_);
}

void FullRectLayer::Backward() {
  if (top_ == NULL) {
    // delta_d = o3 - yLayer* bottom
    hipMemcpy(delta_, output_, output_size_*sizeof(float), hipMemcpyDeviceToDevice);
    AddElement(delta_, output_size_, action_, -y_);
  }

  float alpha = 1.0;
  float beta = 0.0;

  hipblasSgemv(handle_, HIPBLAS_OP_N, input_size_*input_num_, output_size_, &alpha,
              weight_, input_size_*input_num_, delta_, 1, &beta, bottom_->delta_, 1);

  RectDiff(bottom_->output_, bottom_->delta_, input_num_*input_size_);
}

void FullRectLayer::GetGradient() {
  Zeros(grad_weight_, input_num_*input_size_*output_size_);        
  Zeros(grad_bias_, output_size_);        

  float alpha = 1.0; 
  float beta = 1.0;
  hipblasSgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, 
              output_size_, input_num_*input_size_, 1,
              &alpha, delta_, output_size_, input_, 1, &beta, grad_weight_, output_size_);  

  hipblasSaxpy(handle_, output_size_, &alpha, delta_, 1, grad_bias_, 1);    

}

void FullRectLayer::Update() {
  float alpha = - nn_para_->learn_rate_;
  hipblasSaxpy(handle_, output_size_, &alpha, grad_bias_, 1, bias_, 1);    
}

void FullRectLayer::WriteWeights(ofstream &file_out) {
  hipMemcpy(weight_cpu_, weight_, input_num_*input_size_*output_size_*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(bias_cpu_, bias_, output_size_*sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < input_num_*input_size_*output_size_; ++i) {
    file_out << weight_[i] << " ";
  }

  for (int i = 0; i < output_size_; ++i) {
    file_out << bias_[i] << " ";
  }
}

void FullRectLayer::ReadWeights(ifstream &file_in) {
  cout << "FullRectLayer read weights" << endl;
  for (int i = 0; i < input_num_*input_size_*output_size_; ++i) {
    file_in >> weight_cpu_[i];
  }

  for (int i = 0; i < output_size_; ++i) {
    file_in >> bias_cpu_[i];
  }

  hipMemcpy(weight_, weight_cpu_, input_num_*input_size_*output_size_*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(bias_, bias_cpu_, output_size_*sizeof(float), hipMemcpyHostToDevice);
}

void FullLinearLayer::Init(Layer* bottom, Layer* top, hiprandGenerator_t gen, hipblasHandle_t handle) {
  if (bottom == NULL) {
    input_ = NULL;      
    input_im_size_ = nn_para_->im_size_;
    input_size_ = input_im_size_*input_im_size_;
    input_num_ = 1;
  } else {
    input_ = bottom->output_;
    input_im_size_ = bottom->output_im_size_;      
    input_size_ = bottom->output_size_;  
    input_num_ = bottom->output_num_;    
  }

  bottom_ = bottom;
  top_ = top;

  gen_ = gen;
  handle_ = handle;

  output_im_size_ = 0;
  output_num_ = 1;    

  hipMalloc(&output_, output_size_*sizeof(float));
  hipMalloc(&delta_, output_size_*sizeof(float));
  hipMalloc(&weight_, input_num_*input_size_*output_size_*sizeof(float));
  hipMalloc(&bias_, output_size_*sizeof(float));
  hipMalloc(&grad_weight_, input_num_*input_size_*output_size_*sizeof(float));
  hipMalloc(&grad_bias_, output_size_*sizeof(float));

  weight_cpu_ = (float*)malloc(input_num_*input_num_*input_size_*output_size_*sizeof(float));
  bias_cpu_ = (float*)malloc(output_size_*sizeof(float));

  RandGauss(weight_, input_num_*input_num_*input_size_*output_size_, 0.001, gen_, handle_);
  Constants(bias_, output_size_, 0);
  Zeros(delta_, output_size_);
  Zeros(output_, output_size_);        
}

void FullLinearLayer::Forward() {
  float alpha = 1.0;
  float beta = 0.0;
  hipblasSgemv(handle_, HIPBLAS_OP_T, input_num_*input_size_, output_size_, &alpha, 
              weight_, input_num_*input_size_, input_, 1, &beta, output_, 1);   

  hipblasSaxpy(handle_, output_size_, &alpha, bias_, 1, output_, 1);    
}

void FullLinearLayer::Backward() {
  if (top_ == NULL) {
    // delta_d = o3 - y 
    hipMemcpy(delta_, output_, output_size_*sizeof(float), hipMemcpyDeviceToDevice);
    AddElement(delta_, output_size_, action_, -y_);
  }

  float alpha = 1.0;
  float beta = 0.0;
  hipblasSgemv(handle_, HIPBLAS_OP_N, input_size_*input_num_, output_size_, &alpha,
              weight_, input_size_*input_num_, delta_, 1, &beta, bottom_->delta_, 1);
}

void FullLinearLayer::GetGradient() {
  Zeros(grad_weight_, input_num_*input_size_*output_size_);        
  Zeros(grad_bias_, output_size_);        

  float alpha = 1.0; 
  float beta = 1.0;
  hipblasSgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, 
              output_size_, input_num_*input_size_, 1,
              &alpha, delta_, output_size_, input_, 1, &beta, grad_weight_, output_size_);  

  hipblasSaxpy(handle_, output_size_, &alpha, delta_, 1, grad_bias_, 1);    

}

void FullLinearLayer::Update() {
  float alpha = - nn_para_->learn_rate_;
  hipblasSaxpy(handle_, input_num_*input_size_*output_size_, &alpha, grad_weight_, 1, weight_, 1);    
  hipblasSaxpy(handle_, output_size_, &alpha, grad_bias_, 1, bias_, 1);    

}

void FullLinearLayer::WriteWeights(ofstream &file_out) {
  hipMemcpy(weight_cpu_, weight_, input_num_*input_size_*output_size_*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(bias_cpu_, bias_, output_size_*sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < input_num_*input_size_*output_size_; ++i) {
    file_out << weight_cpu_[i] << " ";
  }

  for (int i = 0; i < output_size_; ++i) {
    file_out << bias_cpu_[i] << " ";
  }

}

void FullLinearLayer::ReadWeights(ifstream &file_in) {
  cout << "FullLinearLayer read weights" << endl;
  for (int i = 0; i < input_num_*input_size_*output_size_; ++i) {
    file_in >> weight_cpu_[i];
  }

  for (int i = 0; i < output_size_; ++i) {
    file_in >> bias_cpu_[i];
  }

  hipMemcpy(weight_, weight_cpu_, input_num_*input_size_*output_size_*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(bias_, bias_cpu_, output_size_*sizeof(float), hipMemcpyHostToDevice);
}
