#include "nn.h"

void NN::Init() {   
  
  hiprandCreateGenerator(&gen_, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen_, 132);
  hipblasCreate(&handle_);

  for (int i = 0; i < nn_para_->layer_para_.size(); ++i) {
    cout << "getting layer: " << i << endl;
    Layer* new_layer = GetLayer(nn_para_, nn_para_->layer_para_[i]);    

    if (new_layer == NULL) {
      cerr << "empty layer" << endl;
    }

    layers_.push_back(new_layer);    
  }

  for (int i = 0; i < nn_para_->layer_para_.size(); ++i) {
    if (i == 0) {    
      layers_[i]->Init(NULL, layers_[i+1], gen_, handle_);  
    } else if (i == nn_para_->layer_para_.size() - 1) {
      layers_[i]->Init(layers_[i-1], NULL, gen_, handle_);  
    } else {      
      layers_[i]->Init(layers_[i-1], layers_[i+1], gen_, handle_);  
      layers_[i]->input_ = layers_[i-1]->output_;

      if (i > 0) { 
        if (layers_[i]->input_ != layers_[i-1]->output_) { 
	        cerr << i << " input != last output" << endl;
        }
      }
    }
  }
  
  memory_size_ = 1000;
  memory_idx_ = 0;
  memory_full_ = false;
  action_memory_ = (int*)malloc(memory_size_*sizeof(int));
  reward_memory_ = (float*)malloc(memory_size_*sizeof(float));
  hipMalloc(&im_memory_, memory_size_*im_size_*im_size_*channel_num_*sizeof(float));
  output_cpu_ = (float*)malloc(action_num_*sizeof(float));
  
  cout << "Network Initialization succeeded" << endl;
}

void NN::AddMemIm(float *im) {
  int offset = im_size_*im_size_*channel_num_;
  hipMemcpy(&im_memory_[memory_idx_ * offset], im, offset*sizeof(float), hipMemcpyHostToDevice);
}

void NN::AddMemActionReward(int action, float reward) {
  action_memory_[memory_idx_] = action;
  reward_memory_[memory_idx_] = reward;
  
  (memory_idx_)++;
  memory_idx_ = memory_idx_ % memory_size_;
}

void NN::Forward(int idx) {
  int offset = im_size_*im_size_*channel_num_;
  layers_[0]->input_ = &im_memory_[idx * offset];

  for (int j = 0; j < layers_.size(); ++j) {        
    layers_[j]->Forward();          

    if (j > 0) {
      if (layers_[j]->input_ != layers_[j-1]->output_) {	 
        cerr << j << " input != last output" << endl;
        exit(1);
      }
    }
  }   
}

void NN::Train() {
  int idx;
  if (memory_full_ == true) {
    idx = rand() % (memory_size_ - 1) + 1;
  } else {
    if (memory_idx_ == 0) {
      return;
    } else {
      idx = rand() % memory_idx_ + 1;
    }
  }

  layers_[layers_.size() - 1]->action_ = action_memory_[idx];

  if (reward_memory_[idx] == -10) {
    layers_[layers_.size() - 1]->y_ = -10;
  } else {
    Forward(idx);
    int max_idx;
    float max_value;
    MaxOutput(layers_[layers_.size() - 1]->output_, action_num_, &max_value, &max_idx, handle_);
    layers_[layers_.size() - 1]->y_ = reward_memory_[idx] + 0.1*max_value;
  }

  Forward(idx - 1);
  for (int j = layers_.size() - 1; j >= 0; --j) {
    layers_[j]->Backward();                
  }      

  for (int j = 0; j < layers_.size(); ++j) {   
    layers_[j]->GetGradient();               
  }

  for (int j = 0; j < layers_.size(); ++j) {   
    layers_[j]->Update();             
  }    
}

void NN::GetOutput() {
  hipMemcpy(output_cpu_, layers_[layers_.size() - 1]->output_, action_num_*sizeof(float), hipMemcpyDeviceToHost);
}


void NN::WriteWeights(const char* file_name) {
  ofstream file_out(file_name);
  for (int i = 0; i < layers_.size(); ++i) {
    layers_[i]->WriteWeights(file_out);
  }

  file_out.close();
}

void NN::ReadWeights(const char* file_name) {
  ifstream file_in(file_name);

  if (!file_in.is_open()) {
    cout << "No weights file to read" << endl;
    return;
  }

  for (int i = 0; i < layers_.size(); ++i) {
    layers_[i]->ReadWeights(file_in);
  }

  file_in.close();
}

