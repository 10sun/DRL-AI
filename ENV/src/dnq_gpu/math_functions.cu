#include "hip/hip_runtime.h"
#include "math_functions.h"

bool IsNan(float *a, int size) {
  bool is_nan = false;

  float *b = (float*) malloc(size*sizeof(float));
  hipMemcpy(b, a, size*sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < size; ++i)
    if (b[i] != b[i])
      is_nan = true;

  return is_nan;
}

void PrintCUDA(float *a, int size) {
  float *b = (float*) malloc(size*sizeof(float));
  hipMemcpy(b, a, size*sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < size; ++i)
    cout << b[i] << " ";
  free(b);
}

__global__ void im2col_gpu_kernel_caffe(const int n, const float* data_im,
                const int height, const int width, const int ksize, const int pad,
                const int stride, const int height_col, const int width_col,
          float* data_col) {
CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    index /= width_col;
    int h_out = index % height_col;
    int channel_in = index / height_col;
    int channel_out = channel_in * ksize * ksize;
    int h_in = h_out * stride - pad;
    int w_in = w_out * stride - pad;
    data_col += (channel_out * height_col + h_out) * width_col + w_out;
    data_im += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < ksize; ++i) {
      for (int j = 0; j < ksize; ++j) {
        int h = h_in + i;
        int w = w_in + j;
        *data_col = (h >= 0 && w >= 0 && h < height && w < width) ?
        data_im[i * width + j] : 0;
        data_col += height_col * width_col;
      }
    }
  }
}

void im2col_gpu(const float* data_im, const int channels,
    const int height, const int width, const int ksize, const int pad,
        const int stride, float* data_col) {
    // We are going to launch channels * height_col * width_col kernels, each
      // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height_col * width_col;
  im2col_gpu_kernel_caffe<<<(num_kernels+255)/256, 256>>>(num_kernels, data_im, height, width, ksize, pad, stride, height_col,
                width_col, data_col);
}

__global__ void col2im_gpu_kernel(const int n, const float* data_col,
                                  const int height, const int width, const int channels, const int ksize,
                                  const int pad, const int stride, const int height_col, const int width_col,
                                  float* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    float val = 0;
    int w = index % width + pad;
    int h = (index / width) % height + pad;
    int c = index / (width * height);
    // compute the start and end of the output
    int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
    int w_col_end = min(w / stride + 1, width_col);
    int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
    int h_col_end = min(h / stride + 1, height_col);
                      // equivalent implementation
    int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
    int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
    int coeff_w_col = (1 - stride * height_col * width_col);
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
      }
    }
    data_im[index] = val;
  }
}

void col2im_gpu(const float* data_col, const int channels,
                const int height, const int width, const int ksize, const int pad,
                const int stride, float* data_im) {
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height * width;
  col2im_gpu_kernel<<<(num_kernels+255)/256, 256>>>(
  num_kernels, data_col, height, width, channels, ksize, pad, stride,
  height_col, width_col, data_im);
}

__global__ void RectKernel(float *x, int size) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < size) {
    x[i] = (x[i] > 0) ? x[i]:0;
  }
}

void Rect(float *x, int size) {
  RectKernel<<<(size+255)/256, 256>>>(x, size);
}

__global__ void RectDiffKernel(float *output, float *delta, int size) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < size) {
    if (output[i] < 0.0) {
      delta[i] = 0.0;
    }
  }
}

void RectDiff(float *output, float *delta, int size) {
  RectDiffKernel<<<(size+255)/256, 256>>>(output, delta, size);
}

__global__ void ConstantsKernel(float *x, int size, float value) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < size) {
    x[i] = value;
  }
}

void Constants(float *x, int size, float value) {
  ConstantsKernel<<<(size+255)/256, 256>>>(x, size, value);
}

__global__ void AddElementKernel(float *x, int idx, float y) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i == idx) {
    x[i] += y;    
  }
}

void AddElement(float *x, int size, int idx, float y) {
  AddElementKernel<<<(size + 255)/256, 256>>>(x, idx, y);
}

void MaxOutput(float *x, int size, float *value, int *idx, hipblasHandle_t handle) {
  // index of hipblasIsamax starts from 1 not 0
  hipblasIsamax(handle, size, x, 1, idx);
   --(*idx);
  hipMemcpy(value, &x[*idx], sizeof(float), hipMemcpyDeviceToHost); 
}

void Zeros(float *x, int n) {  
  hipMemset(x, 0, n*sizeof(float));
}

void RandUniform(float *x, int size, float value, hiprandGenerator_t gen) {
  hiprandGenerateUniform(gen, x, size);
}

void RandGauss(float *x, int size, float value, hiprandGenerator_t gen, hipblasHandle_t handle) {    
  hiprandGenerateNormal(gen, x, size, 0, 1);
  hipblasSscal(handle, size, &value, x, 1);
}